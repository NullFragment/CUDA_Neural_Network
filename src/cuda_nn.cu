// Compile using nvcc <file> -lcublas -o <output>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Define block size for thread allocation
#define NUM_THREADS 32 // 32 is max for N^2 threads: 32*32 = 1024

//======================================================================================================================
//=== Structure definitions
//======================================================================================================================

typedef struct _kernelParams
{
    int block_size;
    int grid_size;
} sKernelParams;

typedef struct _matrixSize // Optional Command-line multiplier for matrix sizes
{
    unsigned int A_height, A_width, B_height, B_width, C_height, C_width;
} MatrixSize;

typedef struct _vSize // Optional Command-line multiplier for matrix sizes
{
    unsigned int len_A, len_B, len_C;
} VectorSize;

//======================================================================================================================
//=== Structure functions
//======================================================================================================================

/**
 * @brief -  sets values of vector size structure
 *
 * @param vector_size - pointer to vector size struct
 * @param len - length of all vectors
 */
void SetVectorSize(VectorSize *vector_size, unsigned int len)
{
    vector_size->len_A = len;
    vector_size->len_B = len;
    vector_size->len_C = len;

    printf("Vector A(%u), Vector B(%u), Vector C(%u)\n",
           vector_size->len_A,
           vector_size->len_B,
           vector_size->len_C);

    if (vector_size->len_A != vector_size->len_B ||
        vector_size->len_B != vector_size->len_C ||
        vector_size->len_C != vector_size->len_A)
    {
        printf("ERROR: Matrix sizes do not match!\n");
        exit(-1);
    }
}

/**
 * @brief -  sets values of matrix size structure
 *
 * @param matrixSize - reference to matrix size struct
 * @param widthA - width of matrix A
 * @param heightA - height of matrix A
 * @param widthB - width of matrix B
 * @param heightB - height of matrix B
 * @param widthC - width of matrix C
 * @param heightC - height of matrix C
 */
void SetMatrixSize(MatrixSize *matrixSize,
                   unsigned int widthA, unsigned int heightA,
                   unsigned int widthB, unsigned int heightB,
                   unsigned int widthC, unsigned int heightC)
{
    matrixSize->A_height = heightA;
    matrixSize->A_width = widthA;
    matrixSize->B_height = heightB;
    matrixSize->B_width = widthB;
    matrixSize->C_height = heightC;
    matrixSize->C_width = widthC;

    printf("Matrix A(%u x %u), Matrix B(%u x %u), Matrix C(%u x %u)\n",
           matrixSize->A_width,
           matrixSize->A_height,
           matrixSize->B_width,
           matrixSize->B_height,
           matrixSize->C_width,
           matrixSize->C_height);

}

//======================================================================================================================
//=== GPU memory initialization functions
//======================================================================================================================

/**
 * @brief - allocates memory on GPU for vectors A, B, and C then copies the values for vector A and B
 *          from host PC onto the device
 *
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param vector_size - reference to vector size structure
 * @param host_vA - pointer to host vector A (with values)
 * @param host_vB - pointer to host vector B (with values)
 * @param dev_A - pointer to vector A device memory reference
 * @param dev_B - pointer to vector B device memory reference
 * @param dev_C - pointer to vector C device memory reference
 */
void VectorInitCUDA(int argc, char **argv, int devID, VectorSize *vector_size, float *host_vA, float *host_vB,
                    float *&dev_A, float *&dev_B, float *&dev_C)
{
    // Assign CUDA variables
    hipError_t err;

    // Assign size variables
    size_t size_A = vector_size->len_A * sizeof(float);
    size_t size_B = vector_size->len_B * sizeof(float);
    size_t size_C = vector_size->len_C * sizeof(float);

    // Allocate memory on GPU
    err = hipMalloc((void **) &dev_A, size_A);
    if (err != hipSuccess) printf("Allocate vector A: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_B, size_B);
    if (err != hipSuccess) printf("Allocate vector B: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_C, size_C);
    if (err != hipSuccess) printf("Allocate vector C: %s\n", hipGetErrorString(err));

    // Copy data from host PC to GPU
    err = hipMemcpy(dev_A, host_vA, size_A, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy vector A to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_B, host_vB, size_B, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy vector B to GPU: %s\n", hipGetErrorString(err));

}

/**
 * @brief - allocates memory on GPU for matrices A, B, and C then copies the values for matrices A, B and C
 *          from host PC onto the device
 *
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to vector size structure
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param dev_matrixA - pointer to matrix A device memory reference
 * @param dev_matrixB - pointer to matrix B device memory reference
 * @param dev_matrixC - pointer to matrix C device memory reference
 */
void MatrixInitCUDA(int argc, char **argv, int &devID, MatrixSize *matrixSize,
                    float *host_matrixA, float *host_matrixB, float *host_matrixC,
                    float *&dev_matrixA, float *&dev_matrixB, float *&dev_matrixC)
{
    // Assign CUDA variables
    hipError_t err;

    // Assign size variables
    size_t matrixA_size = matrixSize->A_height * matrixSize->A_width * sizeof(float);
    size_t matrixB_size = matrixSize->B_height * matrixSize->B_width * sizeof(float);
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Allocate memory on GPU
    err = hipMalloc((void **) &dev_matrixA, matrixA_size);
    if (err != hipSuccess) printf("Allocate matrix A: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_matrixB, matrixB_size);
    if (err != hipSuccess) printf("Allocate matrix B: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_matrixC, matrixC_size);
    if (err != hipSuccess) printf("Allocate matrix C: %s\n", hipGetErrorString(err));

    // Copy data from host PC to GPU
    err = hipMemcpy(dev_matrixA, host_matrixA, matrixA_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy matrix A to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_matrixB, host_matrixB, matrixB_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy matrix B to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_matrixC, host_matrixC, matrixC_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy matrix C to GPU: %s\n", hipGetErrorString(err));
}

//======================================================================================================================
//=== CUDA Vector Kernels
//======================================================================================================================
/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of a vector addition
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param dev_vecC - pointer to device memory for vector C
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorAdditionKernel(float *dev_vecA, float *dev_vecB, float *dev_vecC,
                                     float alpha, float beta, int vecLen)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen)
    {
        dev_vecC[i] = alpha * dev_vecA[i] + beta * dev_vecB[i];
    }
}

/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of a vector hadamard product
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param dev_vecC - pointer to device memory for vector C
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorHadamardKernel(float *dev_vecA, float *dev_vecB, float *dev_vecC,
                                     float alpha, float beta, int vecLen)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen)
    {
        dev_vecC[i] = alpha * dev_vecA[i] * beta * dev_vecB[i];
    }
}

/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 *           REMEMBER: Call kernel using: <<<grid, threads, vecLen>>>
 * @brief - kernel for GPU computation of a vector dot product
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param result - pointer to a single float value where the result will be returned
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorDotProduct(float *dev_vecA, float *dev_vecB, float *result,
                                 float alpha, float beta, int vecLen)
{
    extern __shared__ float temp[];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen)
    {
        temp[i] = alpha * dev_vecA[i] * beta * dev_vecB[i];
    }
    __syncthreads();
    if (threadIdx.x == 0)
    {
        float sum = 0.0;
        for (int j = 0; j < vecLen; j++)
        {
            sum += temp[j];
        }
        result[0] = sum;
    }
}

/**
 * @required INPUT AND OUTPUT VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of the vector sigmoid function
 * @param dev_matrixA - pointer to device memory for vector A
 * @param dev_matrixC - pointer to device memory for vector C
 * @param vecLen - length of all vectors
 */
__global__ void VectorSigmoid(float *dev_vecA, float *dev_vecC, int vecLen)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < vecLen)
    {
        float exp = 1 + expf(-dev_vecA[index]);
        dev_vecC[index] = 1 / exp;
    }
}

/**
 * @required INPUT AND OUTPUT VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of the vector sigmoid derivative function
 * @param dev_matrixA - pointer to device memory for vector A
 * @param dev_matrixC - pointer to device memory for vector C
 * @param vecLen - length of all vectors
 */
__global__ void VectorSigmoidDerivative(float *dev_vecA, float *dev_vecC, int vecLen)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < vecLen)
    {
        float exp = 1 + expf(-dev_vecA[index]);
        float sig = 1/exp;
        dev_vecC[index] = sig*(1-sig);
    }
}

//======================================================================================================================
//=== CUDA Vector Kernel Drivers
//======================================================================================================================

/**
 * @brief driver function for computing vector operations
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param vectorSize - reference to vector size structure
 * @param operation - switch-case value for which matrix operation to perform
 *                    1: Vector addition
 *                    2: Vector Hadamard product
 *                    3: Vector dot product
 *                    4: Vector sigmoid function
 *                    5: Vector sigmoid derivative
 * @param host_vectorA - pointer to host vector A (with values)
 * @param host_vectorB - pointer to host vector B (with values)
 * @param host_vectorC - pointer to host vector C (with values)
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 */
void RunVectorKernel(int argc, char **argv, int &devID, VectorSize *vectorSize, int operation,
                     float *host_vectorA, float *host_vectorB, float *host_vectorC, float alpha, float beta)
{
    // Assign CUDA variables
    hipError_t err;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    int gridX = (int) ceil((float) vectorSize->len_C / (float) threads.x);
    int gridY = (int) ceil((float) vectorSize->len_C / (float) threads.y);
    dim3 grid((unsigned int) gridX, (unsigned int) gridY);

    // Assign computation variables
    float *dev_vectorA = NULL;
    float *dev_vectorB = NULL;
    float *dev_vectorC = NULL;

    size_t vectorC_size = vectorSize->len_C * sizeof(float);

    // Initialize memory on GPU
    VectorInitCUDA(argc, argv, devID, vectorSize, host_vectorA, host_vectorB, dev_vectorA, dev_vectorB, dev_vectorC);

    switch (operation)
    {
        case 1:
        {
            // Compute vector addition
            VectorAdditionKernel<<<grid, threads>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta,
                    vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Add Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 2:
        {
            // Compute vector Hadamard Product
            VectorHadamardKernel<<<grid, threads>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta,
                    vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Hadamard Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 3:
        {
            // Compute vector dot product
            VectorDotProduct<<<grid, threads, vectorSize->len_C>>>
                                              (dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Dot product Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 4:
        {
            // Compute sigmoid function
            VectorSigmoid<<<grid, threads>>>(dev_vectorA, dev_vectorC, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Sigmoid Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 5:
        {
            // Compute sigmoid derivative
            VectorSigmoidDerivative<<<grid, threads>>>(dev_vectorA, dev_vectorC, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Sigmoid Derivative Computation: %s\n", hipGetErrorString(err));
            break;
        }
        default:
        {
            printf("ERROR: No vector kernel selected. Operation Aborted");
            break;
        }
    }

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize vector kernel calc: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_vectorC, dev_vectorC, vectorC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        printf("Copy vector C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_vectorA);
    if (err != hipSuccess) printf("Free vector A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_vectorB);
    if (err != hipSuccess) printf("Free vector B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_vectorC);
    if (err != hipSuccess) printf("Free vector C on GPU: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize vector kernel end: %s\n", hipGetErrorString(err));
    printf("Vector Kernel finished.\n");
}

//======================================================================================================================
//=== CUDA Matrix Kernels
//======================================================================================================================

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of matrix additions
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixB - pointer to device memory for matrix B
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixAddKernel(float *dev_matrixA, float *dev_matrixB, float *dev_matrixC,
                                float alpha, float beta, int matrix_width, int matrix_height)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height)
    {
        dev_matrixC[index] = alpha * dev_matrixA[index] + beta * dev_matrixB[index];
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for actual GPU computation for the matrix Hadamard product
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixB - pointer to device memory for matrix B
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixHadamardKernel(float *dev_matrixA, float *dev_matrixB, float *dev_matrixC,
                                     float alpha, float beta, int matrix_width, int matrix_height)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height)
    {
        dev_matrixC[index] = alpha * dev_matrixA[index] * beta * dev_matrixB[index];
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of matrix sigmoid function
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixSigmoid(float *dev_matrixA, float *dev_matrixC,
                              int matrix_width, int matrix_height)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height)
    {
        float exp = 1 + expf(-dev_matrixA[index]);
        dev_matrixC[index] = 1 / exp;
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of the matrix sigmoid derivative function
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixSigmoidDerivative(float *dev_matrixA, float *dev_matrixC,
                                        int matrix_width, int matrix_height)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height)
    {
        float exp = 1 + expf(-dev_matrixA[index]);
        float sig = 1/exp;
        dev_matrixC[index] = sig*(1-sig);
    }
}

//======================================================================================================================
//=== CUDA Matrix Kernel Drivers
//======================================================================================================================

/**
 * @brief -  Uses CUBLAS library to perform alpha(A x B) + beta(C) matrix multiplication and addition
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to vector size structure
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param alpha - value for alpha in CUBLAS function
 * @param beta - value for beta in CUBLAS function
 * @param transposeA - true if A should be transposed
 * @param transposeB - true if B should be transposed
 */

void MatrixMultiplyCUBLAS(int argc, char **argv, int &devID, MatrixSize *matrixSize,
                          float *host_matrixA, float *host_matrixB, float *host_matrixC,
                          float alpha, float beta, bool transposeA, bool transposeB)
{
    // Assign CUDA variables
    hipblasHandle_t handle;
    hipError_t err;
    hipblasCreate(&handle);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devID);
    dim3 threads(NUM_THREADS, NUM_THREADS);
    dim3 grid(matrixSize->C_width / threads.x, matrixSize->C_height / threads.y);

    // Assign computation variables
    float *dev_matrixA = NULL, *dev_matrixB = NULL, *dev_matrixC = NULL;
    int m = matrixSize->A_height;
    int n = matrixSize->B_width;
    int k = matrixSize->A_width;
    hipblasOperation_t transA = HIPBLAS_OP_N, transB = HIPBLAS_OP_N;
    if (transposeA) transA = HIPBLAS_OP_T;
    if (transposeB) transB = HIPBLAS_OP_T;
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Initialize memory on GPU
    MatrixInitCUDA(argc, argv, devID, matrixSize,
                   host_matrixA, host_matrixB, host_matrixC,
                   dev_matrixA, dev_matrixB, dev_matrixC);

    // Perform matrix multiplication
    // SGEMM PARAMS: (handle, transposeA, transposeB, m, n, k, alpha, matrix A, k, matrix B, n, beta, matrix C, n)
    hipblasSgemm(handle, transA, transB, m, n, k, &alpha, dev_matrixA, m,
                dev_matrixB, n, &beta, dev_matrixC, m);
    err = hipGetLastError();
    if (err != hipSuccess) printf("SGEMM: %s\n", hipGetErrorString(err));

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize SGEMM calculation: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_matrixC, dev_matrixC, matrixC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("Copy matrix C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_matrixA);
    if (err != hipSuccess) printf("Free matrix A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixB);
    if (err != hipSuccess) printf("Free matrix B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixC);
    if (err != hipSuccess) printf("Free matrix C on GPU: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize SGEMM end: %s\n", hipGetErrorString(err));
    printf("Matrix Kernel finished.\n");
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief driver function for computing the matrix operations
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to matrix size structure
 * @param operation - switch-case value for which matrix operation to perform
 *                    1: Matrix addition
 *                    2: Matrix Hadamard product
 *                    3: Sigmoid function
 *                    4: Sigmoid derivative
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 */
void RunMatrixKernel(int argc, char **argv, int &devID, MatrixSize *matrixSize, int operation,
                     float *host_matrixA, float *host_matrixB, float *host_matrixC, float alpha, float beta)
{
    // Assign CUDA variables
    hipError_t err;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    int gridX = (int) ceil((float) matrixSize->C_width / (float) threads.x);
    int gridY = (int) ceil((float) matrixSize->C_height / (float) threads.y);
    dim3 grid((unsigned int) gridX, (unsigned int) gridY);

    // Assign computation variables
    float *dev_matrixA = NULL, *dev_matrixB = NULL, *dev_matrixC = NULL;
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Initialize memory on GPU
    MatrixInitCUDA(argc, argv, devID, matrixSize,
                   host_matrixA, host_matrixB, host_matrixC,
                   dev_matrixA, dev_matrixB, dev_matrixC);

    switch (operation)
    {
        case 1:
        {
            // Compute Matrix Addition
            MatrixAddKernel<<<grid, threads>>>(dev_matrixA, dev_matrixB, dev_matrixC, alpha, beta,
                    matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Matrix Add Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 2:
        {
            // Compute Hadamard Product
            MatrixHadamardKernel<<<grid, threads>>>(dev_matrixA, dev_matrixB, dev_matrixC, alpha, beta,
                    matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Matrix Hadamard Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 3:
        {
            // Compute Sigmoid function
            MatrixSigmoid<<<grid, threads>>>(dev_matrixA, dev_matrixC, matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Matrix Sigmoid Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 4:
        {
            // Compute Sigmoid derivative function
            MatrixSigmoidDerivative<<<grid, threads>>>(dev_matrixA, dev_matrixC, matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Matrix Sigmoid Derivative Computation: %s\n", hipGetErrorString(err));
            break;
        }
        default:
        {
            printf("ERROR: No matrix kernel selected. Operation Aborted");
            break;
        }
    }

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device matrix kernel synchronize: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_matrixC, dev_matrixC, matrixC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("Copy matrix C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_matrixA);
    if (err != hipSuccess) printf("Free matrix A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixB);
    if (err != hipSuccess) printf("Free matrix B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixC);
    if (err != hipSuccess) printf("Free matrix C on GPU: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize matrix kernel end: %s\n", hipGetErrorString(err));
    printf("Matrix Kernel finished.\n");

}

//======================================================================================================================
//=== Test Function
//======================================================================================================================

void runTest(int argc, char **argv, int devID)
{
    int N = 10;
    float *host_A, *host_B, *host_C, *host_D;
    float *host_vA, *host_vB, *host_vC, *host_vD, *host_vE;

    MatrixSize *testMatrixSize = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    size_t calcSize = N * N * sizeof(float);
    host_A = (float *) calloc(calcSize, 1);
    host_B = (float *) calloc(calcSize, 1);
    host_C = (float *) calloc(calcSize, 1);
    host_D = (float *) calloc(calcSize, 1);
    SetMatrixSize(testMatrixSize, N, N, N, N, N, N);

    VectorSize *testVectorSize = (VectorSize *) calloc(sizeof(VectorSize), 1);
    size_t calcSize_V = N * sizeof(float);
    host_vA = (float *) calloc(calcSize_V, 1);
    host_vB = (float *) calloc(calcSize_V, 1);
    host_vC = (float *) calloc(calcSize_V, 1);
    host_vD = (float *) calloc(calcSize_V, 1);
    host_vE = (float *) calloc(calcSize_V, 1);
    SetVectorSize(testVectorSize, N);

    for (int i = 0; i < N * N; i++)
    {
        host_A[i] = (float) i;
        host_B[i] = (float) i;
    }

    for (int i = 0; i < N; i++)
    {
        host_vA[i] = (float) i;
        host_vB[i] = (float) i;
    }

    printf("Matrix A:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.0f ", host_A[i * j]);
        }
        printf("\n");
    }
    printf("\nMatrix B:\n");

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.0f ", host_B[i * j]);
        }
        printf("\n");
    }

    printf("Vector A:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vA[i]);
    }
    printf("\n");

    printf("\nVector B:\n");

    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vB[i]);
    }
    printf("\n");

    RunMatrixKernel(argc, argv, devID, testMatrixSize, 3, host_A, host_B, host_C, 1.0, 1.0);
    RunMatrixKernel(argc, argv, devID, testMatrixSize, 4, host_A, host_B, host_D, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 3, host_vA, host_vB, host_vC, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 4, host_vA, host_vB, host_vD, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 5, host_vA, host_vB, host_vE, 1.0, 1.0);

    printf("\nMatrix C:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.10f ", host_C[i * j]);
        }
        printf("\n");
    }
    printf("\nMatrix D:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.10f ", host_D[i * j]);
        }
        printf("\n");
    }

    printf("Vector C:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vC[i]);
    }
    printf("\n");

    printf("\nVector D:\n");

    for (int i = 0; i < N; i++)
    {
        printf("%6.10f ", host_vD[i]);
    }
    printf("\n");

    printf("\nVector E:\n");

    for (int i = 0; i < N; i++)
    {
        printf("%6.10f ", host_vE[i]);
    }
    printf("\n");
}

//======================================================================================================================
//=== Main Function
//======================================================================================================================

/**
 * @brief computes weight matrices for a shallow neural network
 * @param argc - from compiler
 * @param argv - from compiler
 * @return 0 if success
 */
int main(int argc, char **argv)
{
    // Assign CUDA variables
    int devID = 0;
    hipGetDevice(&devID);
    hipError_t mainErr;
//    runTest(argc, argv, devID);

    // Define NN layer lengths
    unsigned int layer_1 = 785;
    unsigned int layer_2 = 129;
    unsigned int layer_3 =  10;

    // Allocate memory for matrices and vectors
    float *a1, *a2, *a3;    // Activation vectors
    float *z2, *z3;         // Pre-sigmoid intermediary vectors
    float *W1, *W2;         // Weight matrices
    float *y;               // One-hot result vector
    float *del3, *del2;     // Error vectors
    float *Del2, *Del1;     // Error gradients

    a1   = (float *) calloc((size_t) layer_1, sizeof(float));
    a2   = (float *) calloc((size_t) layer_2, sizeof(float));
    a3   = (float *) calloc((size_t) layer_3, sizeof(float));
    z2   = (float *) calloc((size_t) layer_2, sizeof(float));
    z3   = (float *) calloc((size_t) layer_3, sizeof(float));
    y    = (float *) calloc((size_t) layer_3, sizeof(float));
    W1   = (float *) calloc((size_t) layer_2*layer_1, sizeof(float));
    W2   = (float *) calloc((size_t) layer_3*layer_2, sizeof(float));
    del3 = (float *) calloc((size_t) layer_3, sizeof(float));
    del2 = (float *) calloc((size_t) layer_2, sizeof(float));
    Del2 = (float *) calloc((size_t) layer_3*layer_2, sizeof(float));
    Del1 = (float *) calloc((size_t) layer_2*layer_1, sizeof(float));

    // Initialize vector and matrix size structures for computation
    MatrixSize *inter2 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *inter3 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *grad1 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *grad2 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *backprop1 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *backprop2 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);

    VectorSize *activation2 = (VectorSize *) calloc(sizeof(VectorSize),1);
    VectorSize *activation3 = (VectorSize *) calloc(sizeof(VectorSize),1);
    VectorSize *delta2 = (VectorSize *) calloc(sizeof(VectorSize),1);
    VectorSize *delta3 = (VectorSize *) calloc(sizeof(VectorSize),1);

    printf("Intermediate 2: ");
    SetMatrixSize(inter2, 1, layer_1, layer_2, layer_1, 1, layer_2);
    printf("Intermediate 3: ");
    SetMatrixSize(inter3, 1, layer_2, layer_3, layer_2, 1, layer_3);
    printf("Grad 1: ");
    SetMatrixSize(grad1, 1, layer_2, 1, layer_1, layer_2, layer_1);
    printf("Grad 2: ");
    SetMatrixSize(grad2, 1, layer_3, 1, layer_2, layer_3, layer_2);
    printf("Backprop 1: ");
    SetMatrixSize(backprop1, layer_2, layer_1, layer_2, layer_1, layer_2, layer_1);
    printf("Backprop 2: ");
    SetMatrixSize(backprop2, layer_3, layer_2, layer_3, layer_2, layer_3, layer_2);

    printf("Activation 2: ");
    SetVectorSize(activation2, layer_2);
    printf("Activation 3: ");
    SetVectorSize(activation3, layer_3);
    printf("Delta 2: ");
    SetVectorSize(delta2, layer_2);
    printf("Delta 3: ");
    SetVectorSize(delta3, layer_3);

    // Set number of epochs and samples
    int epochs = 1; // Number of training epochs (iterations through data)
    int m = 1;   // Number of samples;

    // Perform neural network training
    for(int epoch = 0; epoch < epochs; epoch++)
    {
        for(int sample = 0; sample < m; sample++)
        {
            // FORWARD PROPOGATION:
            //read a1 from file
            MatrixMultiplyCUBLAS(argc, argv, devID, inter2, a1, W1, z2, 1.0, 1.0, false, true); // Compute z2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("z2 Computation: %s\n", hipGetErrorString(mainErr));
            RunVectorKernel(argc, argv, devID, activation2, 4, z2, z2, a2, 1.0, 1.0);           // Compute a2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("a2 Computation: %s\n", hipGetErrorString(mainErr));
            MatrixMultiplyCUBLAS(argc, argv, devID, inter3, a2, W2, z3, 1.0, 1.0, false, true); // Compute z3
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("z3 Computation: %s\n", hipGetErrorString(mainErr));
            RunVectorKernel(argc, argv, devID, activation3, 4, z3, z3, a3, 1.0, 1.0);           // Compute a3
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("a3 Computation: %s\n", hipGetErrorString(mainErr));

            // BACKWARD PROPOGATION:
            RunVectorKernel(argc, argv, devID, delta3, 1, z3, y, del3, 1.0, (float) -1.0);           // Compute del3
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("del3 Computation: %s\n", hipGetErrorString(mainErr));
            MatrixMultiplyCUBLAS(argc, argv, devID, inter3, del3, W2, del2, 1.0, 1.0, false, false); // Compute pre-del2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("pre-del2 Computation: %s\n", hipGetErrorString(mainErr));
            RunVectorKernel(argc, argv, devID, delta2, 5, del2, y, del3, 1.0, (float) -1.0);         // Compute del2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("del2 Computation: %s\n", hipGetErrorString(mainErr));
            MatrixMultiplyCUBLAS(argc, argv, devID, grad1, del2, a1, Del1, 1.0, 1.0, true, false);   // Compute Del1
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("Del1 Computation: %s\n", hipGetErrorString(mainErr));
            MatrixMultiplyCUBLAS(argc, argv, devID, grad2, del3, a2, Del2, 1.0, 1.0, true, false);   // Compute Del2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) printf("Del2 Computation: %s\n", hipGetErrorString(mainErr));

            // Gradient descent
            RunMatrixKernel(argc, argv, devID, backprop1, 1, W1, Del1, W1, 1.0, (float)-1.0/(float)m); // Compute new W1
            RunMatrixKernel(argc, argv, devID, backprop2, 1, W2, Del2, W2, 1.0, (float)-1.0/(float)m); // Compute new W2
        }
    }


    return 0;
}